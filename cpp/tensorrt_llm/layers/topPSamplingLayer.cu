#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2024, NVIDIA CORPORATION.  All rights reserved.
 * Copyright (c) 2021, NAVER Corp.  Authored by CLOVA.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "tensorrt_llm/common/logger.h"
#include "tensorrt_llm/kernels/decodingCommon.h"
#include "tensorrt_llm/kernels/samplingTopPKernels.h"
#include "tensorrt_llm/layers/defaultDecodingParams.h"
#include "tensorrt_llm/layers/layerUtils.h"
#include "topPSamplingLayer.h"

#include <algorithm>
#include <float.h>

using namespace tensorrt_llm::common;
using namespace tensorrt_llm::kernels;
using namespace tensorrt_llm::runtime;

namespace tensorrt_llm::layers
{

static __global__ void setTopPRuntimeArgs(SizeType32 batchSize, SizeType32 topK, SizeType32* topKs,
    SizeType32 topKsSize, float topP, float* topPs, SizeType32 topPsSize, bool* skipDecode,
    SizeType32 const* batchSlots, float* initialTopPBuf)
{
    /**
     * @brief Setup the runtime arguments for topp, broadcasting top_p to top_ps
              and top_k to top_ks.
     */

    auto index = static_cast<SizeType32>(blockIdx.x * blockDim.x + threadIdx.x);
    for (SizeType32 bi = index; bi < batchSize; bi += static_cast<SizeType32>(gridDim.x * blockDim.x))
    {
        auto const batchSlot = batchSlots[bi];
        auto k = topKsSize > 1 ? topKs[batchSlot] : topK;
        auto const p = topPsSize > 1 ? topPs[batchSlot] : topP;
        if (k == 0 && p == 0.0f)
        {
            // TensorRT-LLM's topp implementation does not support topp = 0.0f, but it
            // equivalent to greedy search. So, we set the topk = 1 as an alternative
            // solution.
            k = 1;
        }
        topKs[batchSlot] = k;
        topPs[batchSlot] = p;
        skipDecode[batchSlot] = k > 0;

        initialTopPBuf[batchSlot] = topPs[batchSlot];
    }
}

template <typename T>
TopPSamplingLayer<T>::TopPSamplingLayer(DecoderDomain const& decoderDomain,
    std::shared_ptr<BufferManager> bufferManager, bool isDeterministic, bool isAirTopP)
    : BaseLayer(decoderDomain, bufferManager)
    , mIsDeterministic(isDeterministic)
    , mIsAirTopP(isAirTopP)
{
    TLLM_LOG_TRACE("%s start", __PRETTY_FUNCTION__);

    int deviceId;
    tc::check_cuda_error(hipGetDevice(&deviceId)); // Get the correct device id
    tc::check_cuda_error(hipGetDeviceProperties(&mDeviceProp, deviceId));

    allocateBuffer(mDecoderDomain.getBatchSize());

    TLLM_LOG_TRACE("%s stop", __PRETTY_FUNCTION__);
}

template <typename T>
void TopPSamplingLayer<T>::allocateBuffer(SizeType32 batchSize)
{
    TLLM_LOG_TRACE("%s start", __PRETTY_FUNCTION__);

    if (mIsAirTopP == false)
    {
        mWorkspaceSize = getTopPWorkspaceSize<T>(batchSize, mDecoderDomain.getVocabSizePadded());
    }
    else
    {
        mWorkspaceSize = getAirTopPWorkspaceSize<T>(batchSize, mDecoderDomain.getVocabSizePadded(), mIsDeterministic);
    }

    auto const batchSizeShape = ITensor::makeShape({batchSize});
    mRuntimeTopKDevice = mBufferManager->gpu(batchSizeShape, TRTDataType<SizeType32>::value);
    mRuntimeTopPDevice = mBufferManager->gpu(batchSizeShape, TRTDataType<float>::value);
    mInitialTopPDevice = mBufferManager->gpu(batchSizeShape, TRTDataType<float>::value);
    mTopPDecayDevice = mBufferManager->gpu(batchSizeShape, TRTDataType<float>::value);
    mTopPMinDevice = mBufferManager->gpu(batchSizeShape, TRTDataType<float>::value);
    mTopPResetIdsDevice = mBufferManager->gpu(batchSizeShape, TRTDataType<TokenIdType>::value);
    mSkipDecodeDevice = mBufferManager->gpu(batchSizeShape, TRTDataType<bool>::value);
    mSkipDecodeHost = mBufferManager->pinnedPool(batchSizeShape, TRTDataType<bool>::value);
    auto skipDecodeHostRange = BufferRange<bool>(*mSkipDecodeHost);
    std::fill(skipDecodeHostRange.begin(), skipDecodeHostRange.end(), true);

    auto workspaceSize = std::max({mRuntimeTopKDevice->getSizeInBytes(), mRuntimeTopPDevice->getSizeInBytes(),
        mInitialTopPDevice->getSizeInBytes(), mTopPDecayDevice->getSizeInBytes(), mTopPMinDevice->getSizeInBytes(),
        mTopPResetIdsDevice->getSizeInBytes(), mSkipDecodeDevice->getSizeInBytes()});
    mSetupWorkspaceDevice = mBufferManager->gpu(workspaceSize);

    TLLM_LOG_TRACE("%s stop", __PRETTY_FUNCTION__);
}

template <typename T>
void TopPSamplingLayer<T>::setup(SizeType32 const batchSize, SizeType32 const beamWidth, BufferConstPtr batchSlots,
    std::shared_ptr<BaseSetupParams> const& baseSetupParams)
{
    TLLM_LOG_TRACE("%s start", __PRETTY_FUNCTION__);

    auto setupParams = std::dynamic_pointer_cast<SamplingSetupParams>(baseSetupParams);

    auto const defaultTopK = DefaultDecodingParams::getTopK();
    auto runtimeTopK = setupParams->runtimeTopK.value_or(std::vector<SizeType32>(batchSize, defaultTopK));
    auto runtimeTopP = setupParams->runtimeTopP.value_or(std::vector<float>{});

    auto const runtimeTopKSize = runtimeTopK.size();
    auto const runtimeTopPSize = runtimeTopP.size();

    auto const defaultTopPDecay = DefaultDecodingParams::getTopPDecay();
    auto decayVec = setupParams->topPDecay.value_or(std::vector<float>(batchSize, defaultTopPDecay));

    auto const defaultTopPMin = DefaultDecodingParams::getTopPMin(); // prevent TopP becoming 0.0
    auto topPMinVec = setupParams->topPMin.value_or(std::vector<float>(batchSize, defaultTopPMin));

    auto const defaultTopPResetId = DefaultDecodingParams::getTopPResetId();
    auto topPResetIdsVec = setupParams->topPResetIds.value_or(std::vector<TokenIdType>(batchSize, defaultTopPResetId));

    auto batchSlotsPtr = bufferCastOrNull<SizeType32>(batchSlots);
    auto skipDecodeHostPtr = bufferCastOrNull<bool>(mSkipDecodeHost);
    if (runtimeTopPSize == 0)
    {
        for (SizeType32 bi = 0; bi < batchSize; ++bi)
        {
            auto bid = bi;
            if (batchSlotsPtr)
            {
                bid = batchSlotsPtr[bi];
            }
            skipDecodeHostPtr[bid] = true;
        }
        auto const batchSize = mDecoderDomain.getBatchSize();
        auto skipDecodeHostSlice = IBuffer::slice(mSkipDecodeHost, 0, batchSize);
        mBufferManager->copy(*skipDecodeHostSlice, *mSkipDecodeDevice);
        return;
    }

    for (auto& topP : runtimeTopP)
    {
        if (topP < 0.f || topP > 1.0f)
        {
            TLLM_LOG_WARNING("TopP (%f) is out of range ([0.0, 1.0f]). Clip to closest number.", topP);
            topP = std::clamp(topP, 0.f, 1.f);
        }
    }

    for (auto& decay : decayVec)
    {
        if (decay <= 0.f || decay > 1.0f)
        {
            TLLM_LOG_WARNING(
                "Decay (%f) is out of range ((0.0, 1.0f]). Change to default (%f).", decay, defaultTopPDecay);
            decay = defaultTopPDecay;
        }
    }

    for (auto& topPMin : topPMinVec)
    {
        if (topPMin <= 0.f || topPMin > 1.0f)
        {
            TLLM_LOG_WARNING(
                "TopP min (%f) is out of range ([0.0, 1.0f]). Change to default (%f).", topPMin, defaultTopPMin);
            topPMin = defaultTopPMin;
        }
    }

    auto const topK = runtimeTopK.at(0);
    auto const topP = runtimeTopP.at(0);

    auto setupWorkspaceDevicePtr = reinterpret_cast<SizeType32*>(mSetupWorkspaceDevice->data());
    auto setupWorkspaceDeviceAsFloatPtr = reinterpret_cast<float*>(setupWorkspaceDevicePtr);
    auto runtimeTopKDevicePtr = bufferCastOrNull<SizeType32>(mRuntimeTopKDevice);
    if (runtimeTopKSize > 1)
    {
        TLLM_CHECK_WITH_INFO(static_cast<SizeType32>(runtimeTopK.size()) == batchSize,
            fmtstr("runtimeTopK.size() (%lu) == batchSize (%d) is not satisfied!", runtimeTopK.size(), batchSize));
        copyToWorkspace(*mBufferManager, runtimeTopK, mSetupWorkspaceDevice);
        invokeScatterDecodingParams(
            setupWorkspaceDevicePtr, runtimeTopKDevicePtr, batchSlotsPtr, batchSize, getStream());
    }
    auto runtimeTopPDevicePtr = bufferCast<float>(*mRuntimeTopPDevice);
    if (runtimeTopPSize > 1)
    {
        TLLM_CHECK_WITH_INFO(static_cast<SizeType32>(runtimeTopP.size()) == batchSize,
            fmtstr("runtimeTopP.size() (%lu) == batchSize (%d) is not satisfied!", runtimeTopP.size(), batchSize));
        copyToWorkspace(*mBufferManager, runtimeTopP, mSetupWorkspaceDevice);
        invokeScatterDecodingParams(
            setupWorkspaceDeviceAsFloatPtr, runtimeTopPDevicePtr, batchSlotsPtr, batchSize, getStream());
    }

    auto fillBuffers = [this, batchSize, batchSlotsPtr](
                           std::string name, auto const& vector, auto deviceTmpBuffer, auto deviceBuffer)
    {
        TLLM_CHECK_WITH_INFO(static_cast<SizeType32>(vector.size()) == batchSize,
            fmtstr("%s.size() (%lu) == batchSize (%d) is not satisfied!", name.c_str(), vector.size(), batchSize));
        cudaAutoCpy(deviceTmpBuffer, vector.data(), batchSize, getStream());
        invokeScatterDecodingParams(deviceTmpBuffer, deviceBuffer, batchSlotsPtr, batchSize, getStream());
    };

    auto topPDecayDevicePtr = bufferCastOrNull<float>(mTopPDecayDevice);
    fillBuffers("topPDecay", decayVec, setupWorkspaceDeviceAsFloatPtr, topPDecayDevicePtr);

    auto topPMinDevicePtr = bufferCastOrNull<float>(mTopPMinDevice);
    fillBuffers("topPMin", topPMinVec, setupWorkspaceDeviceAsFloatPtr, topPMinDevicePtr);

    auto topPRestIdsDevicePtr = bufferCastOrNull<SizeType32>(mTopPResetIdsDevice);
    fillBuffers("topPResetIds", topPResetIdsVec, setupWorkspaceDevicePtr, topPRestIdsDevicePtr);

    {
        auto skipDecodeDevicePtr = bufferCastOrNull<bool>(mSkipDecodeDevice);
        auto initialTopPDevicePtr = bufferCastOrNull<float>(mInitialTopPDevice);
        dim3 block(std::min(static_cast<uint32_t>(batchSize), 256u));
        dim3 grid(divUp(static_cast<uint32_t>(batchSize), block.x));
        setTopPRuntimeArgs<<<grid, block, 0, getStream()>>>(batchSize, topK, runtimeTopKDevicePtr, runtimeTopKSize,
            topP, runtimeTopPDevicePtr, runtimeTopPSize, skipDecodeDevicePtr, batchSlotsPtr, initialTopPDevicePtr);
        sync_check_cuda_error();
    }

    auto const skipHostDecodeDeviceSlice = ITensor::slice(mSkipDecodeDevice, 0, mDecoderDomain.getBatchSize());
    auto skipDecodeHostSlice = ITensor::slice(mSkipDecodeHost, 0, mDecoderDomain.getBatchSize());
    mBufferManager->copy(*skipHostDecodeDeviceSlice, *skipDecodeHostSlice);

    if (mIsAirTopP)
    {
        auto smCnt = mDeviceProp.multiProcessorCount;
        if (smCnt <= 0)
        {
            int deviceId;
            check_cuda_error(hipGetDevice(&deviceId)); // Get the correct device id
            hipDeviceProp_t prop;
            check_cuda_error(hipGetDeviceProperties(&prop, deviceId));
            smCnt = prop.multiProcessorCount;
        }
        mAirTopPBlockNum
            = calcAirTopPBlockNum<T>(batchSize, (int) mDecoderDomain.getVocabSizePadded(), smCnt, mIsDeterministic);
    }

    TLLM_LOG_TRACE("%s stop", __PRETTY_FUNCTION__);
}

template <typename T>
void TopPSamplingLayer<T>::forwardAsync(
    std::shared_ptr<BaseDecodingOutputs> const& outputs, std::shared_ptr<BaseDecodingInputs> const& baseInputs)
{
    TLLM_LOG_TRACE("%s start", __PRETTY_FUNCTION__);

    auto inputs = std::dynamic_pointer_cast<SamplingInputs>(baseInputs);

    auto const batchSize = inputs->logits.value()->getDimension<0>();

    auto skipDecodeHostPtr = bufferCastOrNull<bool>(mSkipDecodeHost);
    auto const skip = allOfBatchSlots(bufferCast<SizeType32>(*inputs->batchSlots), skipDecodeHostPtr, batchSize, true);
    if (skip)
    {
        return;
    }

    // Probabilities must be already computed instead of logits
    auto probs = bufferCastOrNull<T>(inputs->logits);
    auto endIds = bufferCastOrNull<TokenIdType>(inputs->endIds);
    auto batchSlots = bufferCast<SizeType32>(*inputs->batchSlots);
    auto curandStatesDevice = inputs->curandStates;
    auto samplingWorkspaceDevice = inputs->samplingWorkspace;

    TLLM_CHECK_WITH_INFO(curandStatesDevice, "No hiprand states provided");
    TLLM_CHECK_WITH_INFO(samplingWorkspaceDevice, "No sampling workspace provided");

    auto finishedInput = (inputs->finished) ? reinterpret_cast<FinishedState const*>(
                             bufferCastOrNull<FinishedState::UnderlyingType>(inputs->finished.value()))
                                            : nullptr;
    auto finishedOutput = (outputs->finished)
        ? reinterpret_cast<FinishedState*>(bufferCastOrNull<FinishedState::UnderlyingType>(outputs->finished.value()))
        : nullptr;

    auto cumLogProbs = bufferCastOrNull<float>(outputs->cumLogProbs);
    auto outputLogProbs = bufferCastOrNull<float>(outputs->outputLogProbsTiled);
    auto sequenceLength = bufferCastOrNull<SizeType32>(outputs->sequenceLength);

    TopPSamplingKernelParams<T> params;
    params.probs = probs;
    params.outputIds = bufferCastOrNull<TokenIdType*>(outputs->outputIdsPtr);
    params.workspace = samplingWorkspaceDevice;
    params.topPs = bufferCastOrNull<float>(mRuntimeTopPDevice);
    params.sequenceLength = sequenceLength;
    params.endIds = endIds;
    params.batchSlots = batchSlots;
    params.finishedInput = finishedInput;
    params.finishedOutput = finishedOutput;
    params.skipDecode = bufferCastOrNull<bool>(mSkipDecodeDevice);
    params.cumLogProbs = cumLogProbs;
    params.outputLogProbs = outputLogProbs;
    params.hiprandState = curandStatesDevice;
    params.batchSize = batchSize;
    params.maxBatchSize = mDecoderDomain.getBatchSize();
    params.vocabSizePadded = mDecoderDomain.getVocabSizePadded();

    if (mIsAirTopP == false)
    {
        invokeBatchTopPSampling<T>(params, getStream());
        sync_check_cuda_error();
    }
    else
    {
        params.blockNum = mAirTopPBlockNum;
        params.isDeterministic = mIsDeterministic;
        invokeBatchAirTopPSampling<T>(params, getStream());
        sync_check_cuda_error();
    }

    sync_check_cuda_error();
    auto runtimeTopPDevicePtr = bufferCastOrNull<float>(mRuntimeTopPDevice);
    auto initialTopPDevicePtr = bufferCastOrNull<float>(mInitialTopPDevice);
    auto topPDecayDevicePtr = bufferCastOrNull<float>(mTopPDecayDevice);
    auto topPMinDevicePtr = bufferCastOrNull<float>(mTopPMinDevice);
    auto topPResetIdsDevice = bufferCastOrNull<TokenIdType>(mTopPResetIdsDevice);
    auto outputIdsPtr = bufferCastOrNull<TokenIdType const*>(outputs->outputIdsPtr);
    invokeComputeToppDecay(runtimeTopPDevicePtr, initialTopPDevicePtr, outputIdsPtr, topPDecayDevicePtr,
        topPMinDevicePtr, topPResetIdsDevice, sequenceLength, batchSlots, batchSize, getStream());
    sync_check_cuda_error();
    TLLM_LOG_TRACE("%s stop", __PRETTY_FUNCTION__);
}

template <typename T>
size_t TopPSamplingLayer<T>::getWorkspaceSize() const noexcept
{
    return mWorkspaceSize;
}

template class TopPSamplingLayer<float>;
template class TopPSamplingLayer<half>;

} // namespace tensorrt_llm::layers
